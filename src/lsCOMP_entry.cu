#include "hip/hip_runtime.h"
#include "lsCOMP_entry.h"
#include "lsCOMP_kernel.h"

// just for debugging, remember to delete later.
#include <stdio.h> 
// Define a macro for error checking
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)
// Function to check CUDA errors
void check(hipError_t result, const char *const func, const char *const file, int const line) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA error at %s:%d code=%d \"%s\" \n", file, line, (unsigned int)result, hipGetErrorString(result));
        // Exit if there is an error
        exit(result);
    }
}


void lsCOMP_compression_uint32_bsize64(uint32_t* d_oriData, unsigned char* d_cmpBytes, size_t* cmpSize, uint3 dims, uint4 quantBins, float poolingTH, hipStream_t stream)
{
    // Data blocking.
    // Treating 3D data as a set of 2D slice, for each slice, we have 8x8 2D blocks.
    uint dimyBlock = (dims.y + 7) / 8;
    uint dimzBlock = (dims.z + 7) / 8;
    uint blockNum = dims.x * dimyBlock * dimzBlock;
    int bsize = 32; // One warp one threadblock for glob sync.
    int gsize = (blockNum + bsize * block_per_thread - 1) / (bsize * block_per_thread);
    int cmpOffSize = gsize + 1;

    // Initializing global memory for GPU compression.
    unsigned int* d_cmpOffset;
    unsigned int* d_locOffset;
    int* d_flag;
    unsigned int glob_sync;
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_locOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_locOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // Compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    lsCOMP_compression_kernel_uint32_bsize64<<<gridSize, blockSize, sizeof(unsigned int)*2, stream>>>(d_oriData, d_cmpBytes, d_cmpOffset, d_locOffset, d_flag, blockNum, dims, quantBins, poolingTH);
    // Check for kernel launch errors
    checkCudaErrors(hipGetLastError());

    // Obtain compression ratio and move data back to CPU.  
    hipMemcpy(&glob_sync, d_cmpOffset+cmpOffSize-1, sizeof(unsigned int), hipMemcpyDeviceToHost);
    *cmpSize = (size_t)glob_sync + blockNum;

    // Free memory that is used.
    hipFree(d_cmpOffset);
    hipFree(d_locOffset);
    hipFree(d_flag);
}

void lsCOMP_decompression_uint32_bsize64(uint32_t* d_decData, unsigned char* d_cmpBytes, size_t cmpSize, uint3 dims, uint4 quantBins, float poolingTH, hipStream_t stream)
{
    // Data blocking.
    // Treating 3D data as a set of 2D slice, for each slice, we have 8x8 2D blocks.
    uint dimyBlock = (dims.y + 7) / 8;
    uint dimzBlock = (dims.z + 7) / 8;
    uint blockNum = dims.x * dimyBlock * dimzBlock;
    int bsize = 32; // One warp one threadblock for glob sync.
    int gsize = (blockNum + bsize * block_per_thread - 1) / (bsize * block_per_thread);
    int cmpOffSize = gsize + 1;

    // Initializing global memory for GPU compression.
    unsigned int* d_cmpOffset;
    unsigned int* d_locOffset;
    int* d_flag;
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_locOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_locOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // Decompression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    lsCOMP_decompression_kernel_uint32_bsize64<<<gridSize, blockSize, sizeof(unsigned int)*2, stream>>>(d_decData, d_cmpBytes, d_cmpOffset, d_locOffset, d_flag, blockNum, dims, quantBins, poolingTH);
    // Check for kernel launch errors
    checkCudaErrors(hipGetLastError());

    // Free memory that is used.
    hipFree(d_cmpOffset);
    hipFree(d_locOffset);
    hipFree(d_flag);
}

void lsCOMP_compression_uint16_bsize64(uint16_t* d_oriData, unsigned char* d_cmpBytes, size_t* cmpSize, uint3 dims, uint4 quantBins, float poolingTH, hipStream_t stream)
{
    // Data blocking.
    // Treating 3D data as a set of 2D slice, for each slice, we have 8x8 2D blocks.
    uint dimyBlock = (dims.y + 7) / 8;
    uint dimzBlock = (dims.z + 7) / 8;
    uint blockNum = dims.x * dimyBlock * dimzBlock;
    int bsize = 32; // One warp one threadblock for glob sync.
    int gsize = (blockNum + bsize * block_per_thread - 1) / (bsize * block_per_thread);
    int cmpOffSize = gsize + 1;

    // Initializing global memory for GPU compression.
    size_t* d_cmpOffset;
    size_t* d_locOffset;
    int* d_flag;
    size_t glob_sync;
    hipMalloc((void**)&d_cmpOffset, sizeof(size_t)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(size_t)*cmpOffSize);
    hipMalloc((void**)&d_locOffset, sizeof(size_t)*cmpOffSize);
    hipMemset(d_locOffset, 0, sizeof(size_t)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // Compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    lsCOMP_compression_kernel_uint16_bsize64<<<gridSize, blockSize, sizeof(size_t)*2, stream>>>(d_oriData, d_cmpBytes, d_cmpOffset, d_locOffset, d_flag, blockNum, dims, quantBins, poolingTH);
    // Check for kernel launch errors
    checkCudaErrors(hipGetLastError());

    // Obtain compression ratio and move data back to CPU.  
    hipMemcpy(&glob_sync, d_cmpOffset+cmpOffSize-1, sizeof(size_t), hipMemcpyDeviceToHost);
    *cmpSize = glob_sync + blockNum;

    // Free memory that is used.
    hipFree(d_cmpOffset);
    hipFree(d_locOffset);
    hipFree(d_flag);
}

void lsCOMP_decompression_uint16_bsize64(uint16_t* d_decData, unsigned char* d_cmpBytes, size_t cmpSize, uint3 dims, uint4 quantBins, float poolingTH, hipStream_t stream)
{
    // Data blocking.
    // Treating 3D data as a set of 2D slice, for each slice, we have 8x8 2D blocks.
    uint dimyBlock = (dims.y + 7) / 8;
    uint dimzBlock = (dims.z + 7) / 8;
    uint blockNum = dims.x * dimyBlock * dimzBlock;
    int bsize = 32; // One warp one threadblock for glob sync.
    int gsize = (blockNum + bsize * block_per_thread - 1) / (bsize * block_per_thread);
    int cmpOffSize = gsize + 1;

    // Initializing global memory for GPU compression.
    size_t* d_cmpOffset;
    size_t* d_locOffset;
    int* d_flag;
    hipMalloc((void**)&d_cmpOffset, sizeof(size_t)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(size_t)*cmpOffSize);
    hipMalloc((void**)&d_locOffset, sizeof(size_t)*cmpOffSize);
    hipMemset(d_locOffset, 0, sizeof(size_t)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // Decompression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    lsCOMP_decompression_kernel_uint16_bsize64<<<gridSize, blockSize, sizeof(size_t)*2, stream>>>(d_decData, d_cmpBytes, d_cmpOffset, d_locOffset, d_flag, blockNum, dims, quantBins, poolingTH);
    // Check for kernel launch errors
    checkCudaErrors(hipGetLastError());

    // Free memory that is used.
    hipFree(d_cmpOffset);
    hipFree(d_locOffset);
    hipFree(d_flag);
}
